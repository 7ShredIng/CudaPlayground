#include "hip/hip_runtime.h"
#include <common.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cpu_bitmap.h>

#define INF 2e10f
#define rnd(x)(x*rand() / RAND_MAX)
#define SPHERES 20
#define DIM 1024

struct Sphere
{
	float r,b,g;
	float radius;
	float x,y,z;
	__device__ float hit(float ox, float oy, float *n)
	{
		float dx = ox - x;
		float dy = oy - y;
		if(dx*dx + dy*dy < radius*radius)
		{
			float dz = sqrtf(radius*radius - dx*dx - dy*dy);
			*n = dz / sqrtf(radius*radius);
			return dz + z;
		}
		return -INF;
	}
};

__device__ __constant__ Sphere sphere[SPHERES];

__global__ void kernel(/*Sphere* sphere, */unsigned char* ptr)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float ox = (x - DIM/2);
	float oy = (y - DIM/2);
	float r = 0, g = 0, b = 0;
	float maxz = -INF;

	for(int i = 0; i < SPHERES; i++)
	{
		float n, distance = sphere[i].hit(ox, oy, &n);
		if(distance > maxz)
		{
			float fscale = n;
			r = sphere[i].r * fscale;
			g = sphere[i].g * fscale;
			b = sphere[i].b * fscale;
			maxz = distance;
		}
	}

	ptr[offset*4 + 0] = (int)(r*255);
	ptr[offset*4 + 1] = (int)(g*255);
	ptr[offset*4 + 2] = (int)(b*255);
	ptr[offset*4 + 3] = 255;
}

struct DataBlock {
    unsigned char* dev_bitmap;
//    Sphere* sphere;
};

int main(void)
{
	DataBlock data;
	//Sphere* sphere;
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	CPUBitmap bitmap(DIM, DIM, &data);
	unsigned char* dev_bitmap;

	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&sphere, sizeof(Sphere)*SPHERES));

	Sphere *temp_sphere = (Sphere*)malloc(sizeof(Sphere)*SPHERES);
	for(int i = 0; i < SPHERES; i++)
	{
		temp_sphere[i].r = rnd(1.0f);
		temp_sphere[i].g = rnd(1.0f);
		temp_sphere[i].b = rnd(1.0f);
		temp_sphere[i].x = rnd(1000.0f) - 500;
		temp_sphere[i].y = rnd(1000.0f) - 500;
		temp_sphere[i].z = rnd(1000.0f) - 500;
		temp_sphere[i].radius = rnd(100.0f) + 20;
	}
	
	//HANDLE_ERROR(hipMemcpy(sphere, temp_sphere, sizeof(Sphere)*SPHERES, hipMemcpyHostToDevice));
	//size_t offset = 0;
	hipError_t e2 = hipMemcpyToSymbol(HIP_SYMBOL(sphere), temp_sphere, sizeof(Sphere)*SPHERES/*, offset, hipMemcpyHostToDevice*/);
	free(temp_sphere);

	dim3 grids(DIM/16, DIM/16);
	dim3 threads(16,16);
	kernel<<<grids,threads>>>(/*sphere, */dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));
	
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Elapsed Time for Sphere Ray Tracing: %3.1f ms\n", elapsedTime);
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	
	bitmap.display_and_exit();

	hipFree(dev_bitmap);
	hipFree(sphere);
}
