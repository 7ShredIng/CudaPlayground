#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "..\..\inc\common.h"
#include "..\..\inc\cpu_anim.h"

#define DIM 1024

struct DataBlock
{
	unsigned char* dev_bitmap;
	CPUAnimBitmap *bitmap;
};

 __global__ void kernel(unsigned char* ptr, int ticks)
 {
	 int x = threadIdx.x + blockIdx.x * blockDim.x;
	 int y = threadIdx.y + blockIdx.y * blockDim.y;
	 int offset = x + y * blockDim.x * gridDim.x;

	 float fx = x - DIM/2;
	 float fy = y - DIM/2;
	 float d = sqrtf((fx * fx) + (fy * fy));
	 
	 unsigned char grey = (unsigned char)(128.0f + (127.0f * cos(d/10.0f - ticks/7.0f) / (d/10.0f + 1.0f)));

	 ptr[offset*4 + 0] = grey;
	 ptr[offset*4 + 1] = grey;
	 ptr[offset*4 + 2] = grey;
	 ptr[offset*4 + 3] = 255;
 }

 void generate_frame(DataBlock* data, int ticks)
{
	dim3 blocks(DIM/16, DIM/16, 1);
	dim3 threads(16, 16, 1);
	kernel<<<blocks,threads>>>(data->dev_bitmap, ticks);
	HANDLE_ERROR(hipMemcpy(data->bitmap->get_ptr(), data->dev_bitmap, data->bitmap->image_size(), hipMemcpyDeviceToHost));
 }

void cleanup(DataBlock *data)
{
	hipFree(data->dev_bitmap);
}

int main(void)
{
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;

	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));
	bitmap.anim_and_exit((void(*)(void*,int))generate_frame, (void(*)(void*))cleanup);
}

