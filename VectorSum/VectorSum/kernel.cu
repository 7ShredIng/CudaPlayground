#include "hip/hip_runtime.h"
#include "common.h"


#define N 5000

__global__ void add(int *a, int *b, int *c)
{
	//int tid = blockIdx.x; //in N blocks
	int tid = threadIdx.x; //in N threads
	if(tid<N)
	{
		c[tid] = a[tid] + b[tid];
	}
}

int main(void)
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));

	for(int i=0; i<N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
	hipEventRecord(start, 0);

	HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));

	add<<<1,N>>>(dev_a, dev_b, dev_c); //in N threads
	//add<<<N,1>>>(dev_a, dev_b, dev_c); //in N blocks

	HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	for(int i=0; i<N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	hipEventElapsedTime(&time, start, stop);
	printf("Time in kernel: %f ms\n", time);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}