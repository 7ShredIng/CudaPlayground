#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include ""
#include <stdlib.h>
#include "..\..\common.h"

__global__ void add(const int value1, const int value2, int *result)
{
	*result = value1 + value2;
}

int main(void)
{
	int result;
	int *dev_result;
	HANDLE_ERROR(hipMalloc((void**)&dev_result, sizeof(int)));

	add<<<1,1>>>(2, 7, dev_result);
	HANDLE_ERROR( hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost));

	printf("result 2 + 7 = %d \n", result);
	HANDLE_ERROR(hipFree(dev_result));

	return 0;
}
